
#include <hip/hip_runtime.h>
#include <stdio.h>

//no return type means that we store all information into the pointers 
__global__ void square(float *d_out, float *d_in) {
    //threadIdx is the index of the thread
    // this is how the thread knows its id.
    int globalId = threadIdx.x;
    float f = d_in[globalId];
    d_out[globalId] = f * f;
}

int main(int argc, char ** argv) {
    const int ARRAY_SIZE = 64;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);
    
    
    
    //this loop initializes an array of ARRAY_SIZE
    float h_in[ARRAY_SIZE];
    for ( int i = 0; i < ARRAY_SIZE; i++ ) {
        h_in[i] = float(i);
    }
    
    float h_out[ARRAY_SIZE];
    
    // declare GPU memory pointers
    // this allows us to know the address of our data on the GPU
    float * d_in;
    float * d_out;
    
    // allocate GPU memory
    hipMalloc((void**) &d_in, ARRAY_BYTES);
    hipMalloc((void**) &d_out, ARRAY_BYTES);
    
    // transfer the array to the GPU
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);
    
    // launch the kernel, node the <<<>>>
    // ARRAY_SIZE indicates the number of cores we want to use.
    square<<<1, ARRAY_SIZE>>>(d_out, d_in);
    
    // transfer the array from GPU to the CPU
    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
    
    //cleaning up memory
    hipFree(d_in);
    hipFree(d_out);
    
    //print the array:
    
    for ( int i = 0; i < ARRAY_SIZE; i++ ) {
        printf("%f", h_out[i]);
        printf( ( ( i%4 ) != 3 ) ? "\t" : "\n");
    }
    
    return 0;
}